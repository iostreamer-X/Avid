#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<conio.h>
#include<string.h>
#include<iostream>

using namespace std;

//The concept is to make a programmable program. Hence the use of these pointers below.
char** name;
double** doubleData;
double*** doubleListData;
int** intData;
int*** intListData;
char** charData;
char*** charListData;
int** boolData;
int*** boolListData;

//How many blocks of each the meta program needs.
int DoubleQuantity,DoubleListX,DoubleListY,IntQuantity,IntListX, IntListY,CharQuantity, CharListX, CharListY, BoolQuantity,BoolListX,BoolListY;

//Total entities in the simulation. 
int total;

//Initializer for the simulation. Creates memory spaces in accordance with the total no. of entities. Uses UVA, hence always use x64. 
int setTotal(int Total){
	total = Total;

	hipHostMalloc((void**)&name, total * sizeof(char*));
	hipHostMalloc((void**)&doubleData, total * sizeof(double*));
	hipHostMalloc((void**)&doubleListData, total * sizeof(double**));
	hipHostMalloc((void**)&intData, total * sizeof(int*));
	hipHostMalloc((void**)&intListData, total * sizeof(int**));
	hipHostMalloc((void**)&charData, total * sizeof(char*));
	hipHostMalloc((void**)&charListData, total * sizeof(char**));
	hipHostMalloc((void**)&boolData, total * sizeof(int*));
	hipHostMalloc((void**)&boolListData, total * sizeof(int**));

	return 1;
}

//Now that the memory spaces are allocated. The meta program memory requirements are taken care of. That is, the memory the program from java needs. 
int setEntity(char* Name,int doubleQuantity, int doubleListX, int doubleListY,
	int intQuantity, int intListX, int intListY,
	int charQuantity, int charListX, int charListY,
	int boolQuantity, int boolListX, int boolListY){
	//This function is called for every entity to be placed and allocated. If you want 1600 enitites. You call this 1600 times. Hence the static index.
	static int index = 0;
	if (index < total){

		hipHostMalloc((void**)&name[index], strlen(Name) * sizeof(char));
		strcpy(name[index],Name);
		if (doubleQuantity > 0)
			hipHostMalloc((void**)&doubleData[index], doubleQuantity*sizeof(double));
		if (doubleListY > 0){
			hipHostMalloc((void**)&doubleListData[index], doubleListY * sizeof(double*));
			for (int i = 0; i < doubleListY; i++)
				hipHostMalloc((void**)&doubleListData[index][i], doubleListX * sizeof(double));
		}

		if (intQuantity > 0)
			hipHostMalloc((void**)&intData[index], intQuantity*sizeof(int));
		if (intListY > 0){
			hipHostMalloc((void**)&intListData[index], intListY * sizeof(int*));
			for (int i = 0; i < intListY; i++)
				hipHostMalloc((void**)&intListData[index][i], intListX * sizeof(int));
		}

		if (charQuantity > 0)
			hipHostMalloc((void**)&charData[index], charQuantity*sizeof(char));
		if (charListY > 0){
			hipHostMalloc((void**)&charListData[index], charListY * sizeof(char*));
			for (int i = 0; i < charListY; i++)
				hipHostMalloc((void**)&charListData[index][i], charListX * sizeof(char));
		}

		if (boolQuantity > 0)
			hipHostMalloc((void**)&boolData[index], boolQuantity*sizeof(int));
		if (boolListY > 0){
			hipHostMalloc((void**)&boolListData[index], boolListY * sizeof(int*));
			for (int i = 0; i < boolListY; i++)
				hipHostMalloc((void**)&boolListData[index][i], boolListX * sizeof(int));
		}
	}
	
	index++;
	return 1;
}

//function pointer for functions modifying doubleData
typedef void(*modifierDouble)(double* array, int index, double operand);

__device__ int strcmp(const char *s1, const char *s2)
{
	for (; *s1 == *s2; s1++, s2++)
		if (*s1 == '\0')
			return 0;
	return ((*(unsigned char *)s1 < *(unsigned char *)s2) ? -1 : +1);
}

/////////////////////////////////////////////////////////////////Double Section


__device__ void addDouble(double* array, int index, double operand){
	array[index] += operand;
}

__device__ void subDouble(double* array, int index, double operand){
	array[index] -= operand;
}

__device__ void mulDouble(double* array, int index, double operand){
	array[index] *= operand;
}

__device__ void divDouble(double* array, int index, double operand){
	array[index] /= operand;
}
__device__ modifierDouble io_addDouble = addDouble;
__device__ modifierDouble io_subDouble = subDouble;
__device__ modifierDouble io_mulDouble = mulDouble;
__device__ modifierDouble io_divDouble = divDouble;

void initDoubleFunctions(modifierDouble* io_modifierDouble){
	hipMemcpyFromSymbol(&io_modifierDouble[0], HIP_SYMBOL(io_addDouble), sizeof(modifierDouble));
	hipMemcpyFromSymbol(&io_modifierDouble[1], HIP_SYMBOL(io_subDouble), sizeof(modifierDouble));
	hipMemcpyFromSymbol(&io_modifierDouble[2], HIP_SYMBOL(io_mulDouble), sizeof(modifierDouble));
	hipMemcpyFromSymbol(&io_modifierDouble[3], HIP_SYMBOL(io_divDouble), sizeof(modifierDouble));
}
/////////////////////////////////////////////////////////////Int Section
__device__ void addInt(int* array, int index, int operand){
	array[index] += operand;
}

__device__ void subInt(int* array, int index, int operand){
	array[index] -= operand;
}

__device__ void mulInt(int* array, int index, int operand){
	array[index] *= operand;
}

__device__ void divInt(int* array, int index, int operand){
	array[index] /= operand;
}

/*
This is the function. I mean, THE function. It simulates and does the core job. For it, the pointers declared on top are required and an instruction set.
The instruction set is what makes this code programmble. Here's the legend for the instruction set:
1)io_name - Name of the Entity
2)io_morphCode - an integer which tells which data structure to modify(doubleData/intData/...)
3)io_functionCode - an integer which tells which function to apply and use(0 means add)
4)io_index1 - an index to the memory of the data structure chosen by io_morphCode. Signifies which trait of the entity to modify.
5)io_index2 - same as io_index1 but for 2D arrays.
6)io_operand - the operand for the function operation.
7)isMemory - tells if the operand lies in the memory or not and if does then which data structure it lies in.
8)entityIndex - The entity whose memory should be used as operand
8)memIndices - the address of that memory operand
*/
__global__ void ultimateCoder(
	char** name,double** doubleData,int** intData,
	char* io_name, int io_morphCode, int io_functionCode, int io_index1, int io_index2, double io_operand, int isMemory, int entityIndex, int memIndex1, int memIndex2,
	modifierDouble* io_modierDouble){
	
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (!strcmp(io_name, name[index])){
		switch (io_morphCode){
		case 0://Double
			switch (isMemory)
			{
			case 0:
				break;
			case 1://operand from Int
				io_operand = intData[entityIndex][memIndex1];
				break;
			case 2://operand from IntList
				//io_operand = intListData[entityIndex][memIndex1][memIndex2];
				break;
			case 3://operand from Double
				io_operand = doubleData[entityIndex][memIndex1];
				break;
			case 4://operand from DoubleList
				//io_operand = doubleListData[entityIndex][memIndex1][memIndex2];
				break;
			}
			io_modierDouble[io_functionCode](doubleData[index], io_index1, io_operand);
			break;
		}
	}

}

int main(){
	setTotal(2);
	setEntity("yay",1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 2, 1);
	setEntity("yay2",1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 2, 1);
	doubleData[0][0]= 19;
	modifierDouble* io_modifierDouble;
	hipHostMalloc((void**)&io_modifierDouble, 4 * sizeof(modifierDouble));
	initDoubleFunctions(io_modifierDouble);
	char* na;
	hipHostMalloc((void**)&na, 3 * sizeof(char));
	strcpy(na, "yay2");
	ultimateCoder <<<1,2>>> (name, doubleData, intData, na, 0, 0, 0, 0, 0, 3, 0, 0, 0, io_modifierDouble);
	hipDeviceSynchronize();
	strcpy(na, "yay");
	ultimateCoder <<<1,2>>> (name, doubleData, intData, na, 0, 1, 0, 0, 3, 0, 0, 0, 0, io_modifierDouble);
	hipDeviceSynchronize();
	printf("%f %f", doubleData[1][0], doubleData[0][0]);
	getch();
}